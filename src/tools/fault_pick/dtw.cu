#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <boost/thread.hpp>

__global__
void dtw ( int n
         , int p_num_samp
         , int win
         , float * A
         , float * B
         , int * toggle
         , float * S
         )
{

  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if ( i < n )
  {

    // init

    int length;
    int num_samp;
    int num_layers;
    int width;
    float ** dist;
    
    num_samp = p_num_samp;
    length = num_samp;
    num_layers = 2*num_samp - 1;
    width = 2*win + 1;
    dist = new float *[3];

    for(int k=0;k<3;k++)
    {
      dist[k] = new float [width];
      for(int i=0;i<width;i++)
      {
        dist[k][i] = 0;
      }
    }
    
    int N=num_layers*width;
    for(int k=0;k<num_layers;k++)
    {
      for(int j=0;j<width;j++)
      {
        toggle[i*N+width*k+j] = 0;
      }
    }
    

    int inter = 2;
    int min_toggle = -1;
    int s1,s2;
    int L,L_1,L_2;
    for ( int l=0; l < num_layers; l++ )
    {
      L = l%3;
      L_1 = (l-1+3)%3;
      L_2 = (l-2+3)%3;
      if ( l % 2 == 0 )
      {
        for ( int w=0; w < width; w++ )
        {
          s1 = -win + (l/2) + w;
          if ( s1 >= 0 && s1 >= (l/2) - win && s1 < num_samp)
          {
            s2 = l - s1;
            if ( s2 >= 0 && s2 >= (l/2) - win && s2 < num_samp)
            {
              min_toggle = -1;
              dist[L][w] = 10000;
              if ( l > 0 )
              {
                if (dist[L_1][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_1][w];
                  min_toggle = 0;
                }
                if ( w > 0 )
                {
                  if (dist[L_1][w-1] <= dist[L][w])
                  {
                    dist[L][w] = dist[L_1][w-1];
                    min_toggle = 1;
                  }
                }
              }
              if ( l > 1 )
              {
                if (dist[L_2][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_2][w];
                  min_toggle = 2;
                }
              }
              if(min_toggle==-1)
              {
                dist[L][w] = 0;
              }
              toggle[i*N+width*l+w] = min_toggle;
              for(int k=-inter;k<=inter;k++)
              {
                int count = 0;
                if (s1+k>=0&&s1+k<length&&s2+k>=0&&s2+k<length)
                {
                count++;
                dist[L][w] += fabs(A[i*p_num_samp+s1+k]-B[i*p_num_samp+s2+k]);//metric(A[s1+k],B[s2+k]);
                }
                dist[L][w] /= count+0.01f;
              }
            }
          }
        }
      }
      else
      {
        for ( int w=0; w < width; w++ )
        {
          s1 = -win + (l/2)+1 + w;
          if ( s1 >= 0 && s1 >= (l/2)+1 - win && s1 < num_samp)
          {
            s2 = l - s1;
            if ( s2 >= 0 && s2 >= (l/2)+1 - win && s2 < num_samp)
            {
              min_toggle = -1;
              dist[L][w] = 10000;
              if ( l > 0 )
              {
                if (dist[L_1][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_1][w];
                  min_toggle = 0;
                }
                if ( w+1 < width )
                {
                  if (dist[L_1][w+1] <= dist[L][w])
                  {
                    dist[L][w] = dist[L_1][w+1];
                    min_toggle = 3;
                  }
                }
              }
              if ( l > 1 )
              {
                if (dist[L_2][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_2][w];
                  min_toggle = 2;
                }
              }
              if (min_toggle==-1)
              {
                dist[L][w] = 0;
              }
              toggle[i*N+width*l+w] = min_toggle;
              for(int k=-inter;k<=inter;k++)
              {
                int count = 0;
                if (s1+k>=0&&s1+k<length&&s2+k>=0&&s2+k<length)
                {
                count++;
                dist[L][w] += fabs(A[i*p_num_samp+s1+k]-B[i*p_num_samp+s2+k]);//metric(A[s1+k],B[s2+k]);
                }
                dist[L][w] /= count+0.01f;
              }
            }
          }
        }
      }
    }
    

    for(int k=0;k<p_num_samp;k++)
    {
      S[i*p_num_samp+k] = 0;
    }
    int S1=1;
    int S2=1;
    int Layer = num_layers-1;
    int Win = win;
    int iter = 0;
    while(S1>0&&S2>0&&iter<2*length)
    {
      iter++;
      S1 = -win + (Layer/2) + (Layer%2) + Win;
      S2 = Layer - S1;
      S[i*p_num_samp+S1] = S2-S1;
      if(toggle[i*N+width*Layer+Win] == 0)
      {
        Layer--;
      }
      else
      if(toggle[i*N+width*Layer+Win] == 1)
      {
        Layer--;
        Win--;
      }
      else
      if(toggle[i*N+width*Layer+Win] == 2)
      {
        Layer-=2;
      }
      else
      if(toggle[i*N+width*Layer+Win] == 3)
      {
        Layer--;
        Win++;
      }
    }


    for(int k=0;k<3;k++)
    {
      delete [] dist[k];
    }
    delete [] dist;

  }

}

void dtw_cpu ( std::vector<int> indices
             , int n
             , int p_num_samp
             , int win
             , float * p_A
             , float * p_B
             , float * p_S
             )
{

  for(int k=0;k<indices.size();k++)
  {
    int i = indices[k];

    // init

    int length;
    int num_samp;
    int num_layers;
    int width;
    float ** dist;
    int ** toggle;

    float * A = &p_A[i*p_num_samp];
    float * B = &p_B[i*p_num_samp];
    float * S = &p_S[i*p_num_samp];
    
    num_samp = p_num_samp;
    length = num_samp;
    num_layers = 2*num_samp - 1;
    width = 2*win + 1;
    dist = new float *[3];
    for(int k=0;k<3;k++)
    {
      dist[k] = new float [width];
      for(int i=0;i<width;i++)
      {
        dist[k][i] = 0;
      }
    }
    toggle = new int *[num_layers];
    for(int k=0;k<num_layers;k++)
    {
      toggle[k] = new int [width];
      for(int i=0;i<width;i++)
      {
        toggle[k][i] = 0;
      }
    }


    int inter = 2;
    int min_toggle = -1;
    int s1,s2;
    int L,L_1,L_2;
    for ( int l=0; l < num_layers; l++ )
    {
      L = l%3;
      L_1 = (l-1+3)%3;
      L_2 = (l-2+3)%3;
      if ( l % 2 == 0 )
      {
        for ( int w=0; w < width; w++ )
        {
          s1 = -win + (l/2) + w;
          if ( s1 >= 0 && s1 >= (l/2) - win && s1 < num_samp)
          {
            s2 = l - s1;
            if ( s2 >= 0 && s2 >= (l/2) - win && s2 < num_samp)
            {
              min_toggle = -1;
              dist[L][w] = 10000;
              if ( l > 0 )
              {
                if (dist[L_1][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_1][w];
                  min_toggle = 0;
                }
                if ( w > 0 )
                {
                  if (dist[L_1][w-1] <= dist[L][w])
                  {
                    dist[L][w] = dist[L_1][w-1];
                    min_toggle = 1;
                  }
                }
              }
              if ( l > 1 )
              {
                if (dist[L_2][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_2][w];
                  min_toggle = 2;
                }
              }
              if(min_toggle==-1)
              {
                dist[L][w] = 0;
              }
              toggle[l][w] = min_toggle;
              for(int k=-inter;k<=inter;k++)
              {
                int count = 0;
                if (s1+k>=0&&s1+k<length&&s2+k>=0&&s2+k<length)
                {
                count++;
                dist[L][w] += fabs(A[s1+k]-B[s2+k]);//metric(A[s1+k],B[s2+k]);
                }
                dist[L][w] /= count+0.01f;
              }
            }
          }
        }
      }
      else
      {
        for ( int w=0; w < width; w++ )
        {
          s1 = -win + (l/2)+1 + w;
          if ( s1 >= 0 && s1 >= (l/2)+1 - win && s1 < num_samp)
          {
            s2 = l - s1;
            if ( s2 >= 0 && s2 >= (l/2)+1 - win && s2 < num_samp)
            {
              min_toggle = -1;
              dist[L][w] = 10000;
              if ( l > 0 )
              {
                if (dist[L_1][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_1][w];
                  min_toggle = 0;
                }
                if ( w+1 < width )
                {
                  if (dist[L_1][w+1] <= dist[L][w])
                  {
                    dist[L][w] = dist[L_1][w+1];
                    min_toggle = 3;
                  }
                }
              }
              if ( l > 1 )
              {
                if (dist[L_2][w] <= dist[L][w])
                {
                  dist[L][w] = dist[L_2][w];
                  min_toggle = 2;
                }
              }
              if (min_toggle==-1)
              {
                dist[L][w] = 0;
              }
              toggle[l][w] = min_toggle;
              for(int k=-inter;k<=inter;k++)
              {
                int count = 0;
                if (s1+k>=0&&s1+k<length&&s2+k>=0&&s2+k<length)
                {
                count++;
                dist[L][w] += fabs(A[s1+k]-B[s2+k]);//metric(A[s1+k],B[s2+k]);
                }
                dist[L][w] /= count+0.01f;
              }
            }
          }
        }
      }
    }
    

    int S1=1;
    int S2=1;
    int Layer = num_layers-1;
    int Win = win;
    int iter = 0;
    for(int k=0;k<p_num_samp;k++)
    {
      S[k] = 0;
    }
    while(S1>0&&S2>0&&iter<2*length)
    {
      iter++;
      S1 = -win + (Layer/2) + (Layer%2) + Win;
      S2 = Layer - S1;
      S[S1] = S2-S1;
      if(toggle[Layer][Win] == 0)
      {
        Layer--;
      }
      else
      if(toggle[Layer][Win] == 1)
      {
        Layer--;
        Win--;
      }
      else
      if(toggle[Layer][Win] == 2)
      {
        Layer-=2;
      }
      else
      if(toggle[Layer][Win] == 3)
      {
        Layer--;
        Win++;
      }
    }


    for(int k=0;k<3;k++)
    {
      delete [] dist[k];
    }
    delete [] dist;
    for(int k=0;k<num_layers;k++)
    {
      delete [] toggle[k];
    }
    delete [] toggle;

  }

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

int main(void)
{


  int nt = 2*1024;
  int nx = 12*200;//1024;
  int win = 100;

  float *A, *B, *S, *S_cpu, *d_A, *d_B, *d_S;
  int *d_toggle;
  A = (float*)malloc(nt*nx*sizeof(float));
  B = (float*)malloc(nt*nx*sizeof(float));
  S = (float*)malloc(nt*nx*sizeof(float));
  S_cpu = (float*)malloc(nt*nx*sizeof(float));
  int num_layers = 2*nt - 1;
  int width = 2*win + 1;
  hipMalloc(&d_A, nt*nx*sizeof(float)); 
  hipMalloc(&d_B, nt*nx*sizeof(float)); 
  hipMalloc(&d_S, nt*nx*sizeof(float)); 
  hipMalloc(&d_toggle, nx*num_layers*width*sizeof(float));

  for(int x=0,k=0;x<nx;x++)
  for(int t=0;t<nt;t++,k++)
  {
    A[k] = (x+1)*t/(float)nt;
    B[k] = (x+1)*(t+(x+(int)sqrt(t)+1)%30)/(float)nt;
    S[k] = 0;
  }

  hipMemcpy(d_A,A,nt*nx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,nt*nx*sizeof(float),hipMemcpyHostToDevice);

  std::cout << "begin GPU" << std::endl;
  clock_t start_gpu = clock();
  dtw<<<(nx+255)/256,256>>>(nx,nt,win,d_A,d_B,d_toggle,d_S);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  clock_t end_gpu = clock();
  std::cout << "end GPU" << std::endl;
  std::cout << "GPU time:" << end_gpu - start_gpu << std::endl;

  hipMemcpy(S,d_S,nt*nx*sizeof(float),hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_S);
  hipFree(d_toggle);

  int nthreads = 8;
  std::vector<boost::thread*> threads;
  std::vector<std::vector<int> > indices(nthreads);
  for(int i=0;i<nx;i++)
  {
    indices[i%nthreads].push_back(i);
  }
  std::cout << "begin CPU" << std::endl;
  clock_t start_cpu = clock();
  for(int i=0;i<nthreads;i++)
  {
    threads.push_back(new boost::thread(dtw_cpu,indices[i],nx,nt,win,A,B,S_cpu));
  }
  for(int i=0;i<nthreads;i++)
  {
    threads[i]->join();
  }
  clock_t end_cpu = clock();
  std::cout << "end CPU" << std::endl;
  std::cout << "CPU time:" << end_cpu - start_cpu << std::endl;
  std::cout << "speedup:" << (float)(end_cpu - start_cpu)/(float)(end_gpu - start_gpu) << "X" << std::endl;
  double Error = 0;
  for(int i=0;i<nt*nx;i++)
  {
    Error += fabs(S[i]-S_cpu[i]);
  }
  std::cout << "Error:" << Error << std::endl;

  return 0;
}

